#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : Jack Weber
 Description   : Project 3
 To build use  : make, ./main
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include "projection.h"
#include "ImageWriter.h"
#include "Packed3DArray.h"

//Function used by the kernel to determine the begining of a sheet. (not always a "sheet"). (z axis)
__device__ size_t getCoord(size_t z, int nRows, int nCols, int nSheets, int pt){
	size_t x_coord = ( blockDim.x * blockIdx.x + threadIdx.x);
	size_t y_coord = ( blockDim.y * blockIdx.y + threadIdx.y);

	if(pt == 1){
		return z * nRows * nCols + x_coord * nCols + y_coord;
	}
	else if(pt == 2){
		x_coord = (nRows - 1) - x_coord;
		return z * nRows * nCols + x_coord * nCols + y_coord;
	}
	else if(pt == 3){
		return x_coord * nSheets * nCols + z * nCols + y_coord;
	}
	else if(pt == 4){
		x_coord = (nSheets - 1) - x_coord;
		return x_coord * nSheets * nCols + z * nCols + y_coord;
	}
	else if(pt == 5){
		return y_coord * nSheets * nCols + x_coord * nSheets + z;
	}
	else if(pt == 6){
		return ((nCols - 1) - y_coord) * nSheets * nCols + x_coord * nSheets + z;
	} else {
		return 0;
	}
}

//Kernel for maz image calculation. fils maxImage with the corresponding data and fils sums with data needed for sumImageCalc
__global__ void maxImageCalc (int nRows, int nCols, int nSheets, unsigned char* buffer, unsigned char* maxImage, float* sums, int pt)
{

	size_t x_coord = ( blockDim.x * blockIdx.x + threadIdx.x);
	size_t y_coord = ( blockDim.y * blockIdx.y + threadIdx.y);

	if(pt == 1 || pt == 2){
		int myID = y_coord * nRows + x_coord;
		maxImage[myID] = 0;
		sums[myID] = 0.0;
		for(int i = 0; i < nSheets; i++){
			if(buffer[getCoord(i, nRows, nCols, nSheets, pt)] > maxImage[myID])
				maxImage[myID] = buffer[getCoord(i, nRows, nCols, nSheets, pt)];
			sums[myID] += (buffer[getCoord(i, nRows, nCols, nSheets, pt)]) * (i+1)/nSheets;
		}
	}
	else if(pt == 3 || pt == 4){
		int myID = y_coord * nRows + x_coord;
		maxImage[myID] = 0;
		for(int i = 0; i < nCols; i++){
			if(buffer[getCoord(i, nRows, nCols, nSheets, pt)] > maxImage[myID])
				maxImage[myID] = buffer[getCoord(i, nRows, nCols, nSheets, pt)];
			sums[myID] += (buffer[getCoord(i, nRows, nCols, nSheets, pt)]) * (i+1)/nSheets;
		}
	}
	else if(pt == 5 || pt == 6){
		int myID = y_coord * nSheets + x_coord;
		maxImage[myID] = 0;
		for(int i = 0; i < nRows; i++){
			if(buffer[getCoord(i, nRows, nCols, nSheets, pt)] > maxImage[myID])
				maxImage[myID] = buffer[getCoord(i, nRows, nCols, nSheets, pt)];
			sums[myID] += (buffer[getCoord(i, nRows, nCols, nSheets, pt)]) * (i+1)/nSheets;
		}
	}
}

//Kernel that computes the sum image data from the data found in the maxImageCalc kernel
__global__ void sumImageCalc(float* sumInfo, float maximum, unsigned char* sumImage, int nRows){
	size_t x_coord = ( blockDim.x * blockIdx.x + threadIdx.x);
	size_t y_coord = ( blockDim.y * blockIdx.y + threadIdx.y);
	int myID = y_coord * nRows + x_coord;
	sumImage[myID] = round((sumInfo[myID]/maximum)*255.0);
}

//Entry point into the program
int main(int argc, char** argv)
{
	//Give arguments to the projection class
    if(!argv[6]){
        std::cout << "Not enough arguments\n";
    }
    Projection* projection = new Projection();
    projection->nRows = atoi(argv[1]);
    projection->nCols = atoi(argv[2]);
    projection->nSheets = atoi(argv[3]);
    projection->filename = argv[4];
    projection->pt = atoi(argv[5]);
    projection->output = argv[6];
    
    //Read in a file for projection class
    projection->readFile();
    char* hostBuffer = projection->stream;
    
	//Declare pointers to be used on device
    unsigned char *d_maxImage, *d_sumImage, *d_buffer;
	float *d_sumWorking;

	//Copy voxel data to the cpu.
	hipMalloc((void**)&d_buffer, projection->size());
    hipMemcpy(d_buffer, hostBuffer, projection->size(), hipMemcpyHostToDevice);
	
	//Determine the size of the images
	hipMalloc((void**)&d_maxImage, projection->imageSize() * sizeof(char));
	hipMalloc((void**)&d_sumImage, projection->imageSize() * sizeof(char));
	hipMalloc((void**)&d_sumWorking, projection->imageSize() * sizeof(float));


	//Create grid sizes, then launch kernel for maxImageCalc
	if(projection->pt == 1 || projection->pt == 2){
		dim3 threadsPerBlock(16, 16);
		dim3 blocksPerGrid(
			(projection->nCols + threadsPerBlock.x - 1)/threadsPerBlock.x,
			(projection->nRows + threadsPerBlock.y - 1)/threadsPerBlock.y);
		maxImageCalc<<<blocksPerGrid, threadsPerBlock>>>(projection->nCols, projection->nRows, projection->nSheets, d_buffer, d_maxImage, d_sumWorking, projection->pt);
	}
	else if(projection->pt == 3 || projection->pt == 4){
		dim3 threadsPerBlock(16, 16);
		dim3 blocksPerGrid(
			(projection->nSheets + threadsPerBlock.x - 1)/threadsPerBlock.x,
			(projection->nRows + threadsPerBlock.y - 1)/threadsPerBlock.y);
		maxImageCalc<<<blocksPerGrid, threadsPerBlock>>>(projection->nSheets, projection->nRows, projection->nCols, d_buffer, d_maxImage, d_sumWorking, projection->pt);
	}
	else if(projection->pt == 5 || projection->pt == 6){
		dim3 threadsPerBlock(16, 16);
		dim3 blocksPerGrid(
			(projection->nCols + threadsPerBlock.x - 1)/threadsPerBlock.x,
			(projection->nSheets + threadsPerBlock.y - 1)/threadsPerBlock.y);
		maxImageCalc<<<blocksPerGrid, threadsPerBlock>>>(projection->nSheets, projection->nRows, projection->nCols, d_buffer, d_maxImage, d_sumWorking, projection->pt);
	}
	else {
		std::cout << "WRONG PT";
	}

	//Get the result
	unsigned char* h_maxImage;
	h_maxImage = new unsigned char[projection->imageSize()];
	hipMemcpy(h_maxImage, d_maxImage, projection->imageSize() * sizeof(char), hipMemcpyDeviceToHost);
	const unsigned char* test = h_maxImage;

	float* h_sums = new float[projection->imageSize()];
	hipMemcpy(h_sums, d_sumWorking, projection->imageSize() * sizeof(float), hipMemcpyDeviceToHost);

	//Calculate the maxs
	float h_sum_maximum = 0.0;
	for(int i = 0; i < projection->imageSize(); i++){
		if(h_sum_maximum < h_sums[i])
			h_sum_maximum = h_sums[i];
	}

	//Create grid sizes, launch the next kernel (sumImageCalc)
	if(projection->pt == 1 || projection->pt == 2){
		dim3 threadsPerBlock(16, 16);
		dim3 blocksPerGrid(
			(projection->nCols + threadsPerBlock.x - 1)/threadsPerBlock.x,
			(projection->nRows + threadsPerBlock.y - 1)/threadsPerBlock.y);
		sumImageCalc<<<blocksPerGrid, threadsPerBlock>>>(d_sumWorking, h_sum_maximum, d_sumImage, projection->nRows);
	}
	else if(projection->pt == 3 || projection->pt == 4){
		dim3 threadsPerBlock(16, 16);
		dim3 blocksPerGrid(
			(projection->nSheets + threadsPerBlock.x - 1)/threadsPerBlock.x,
			(projection->nRows + threadsPerBlock.y - 1)/threadsPerBlock.y);
		sumImageCalc<<<blocksPerGrid, threadsPerBlock>>>(d_sumWorking, h_sum_maximum, d_sumImage, projection->nRows);
	}
	else if(projection->pt == 5 || projection->pt == 6){
		dim3 threadsPerBlock(16, 16);
		dim3 blocksPerGrid(
			(projection->nCols + threadsPerBlock.x - 1)/threadsPerBlock.x,
			(projection->nSheets + threadsPerBlock.y - 1)/threadsPerBlock.y);
		sumImageCalc<<<blocksPerGrid, threadsPerBlock>>>(d_sumWorking, h_sum_maximum, d_sumImage, projection->nSheets);
	}
	else {
		std::cout << "WRONG PT";
	}

	//Get the results of the sumImageCalc kernel
	unsigned char* h_sumImage = new unsigned char[projection->imageSize()];
	hipMemcpy(h_sumImage, d_sumImage, projection->imageSize() * sizeof(char), hipMemcpyDeviceToHost);
	const unsigned char* f_h_sumImage = h_sumImage;

	//Free the memory on device.
	hipFree(d_maxImage);
	hipFree(d_sumImage);
	hipFree(d_buffer);
	hipFree(d_sumWorking);
	hipDeviceSynchronize();


	//Print the images.
	std::string maxName = argv[6];
	std::string sumName = argv[6];
	maxName.append("MAX.png");
	sumName.append("SUM.png");

	if(projection->pt == 1 || projection->pt == 2){
		projection->writeTheFile(maxName, projection->nCols, projection->nRows, test);
		projection->writeTheFile(sumName, projection->nCols, projection->nRows, f_h_sumImage);
	}
	else if(projection->pt == 3 || projection->pt == 4){
		projection->writeTheFile(maxName, projection->nSheets, projection->nRows, test);
		projection->writeTheFile(sumName, projection->nSheets, projection->nRows, f_h_sumImage);
	}
	else if(projection->pt == 5 || projection->pt == 6){
		projection->writeTheFile(maxName, projection->nCols, projection->nSheets, test);
		projection->writeTheFile(sumName, projection->nCols, projection->nSheets, f_h_sumImage);
	}
	else {
		std::cout << "WRONG PT 2\n";
	}
	

	return 0;
}

